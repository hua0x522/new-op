#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "reorder_map_cuda.h"
#include <cstdio>

#define cdiv(x, y) (((x) + (y) - 1) / (y))

__global__ void reorder_map_kernel(int* out_in_map, int* loc_map, int* reduced_map, int n_points) {
    __shared__ int shm_map[64];
    __shared__ int cnt;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    if (tid == 0 && bid == 0) {
        printf("out_in_map: %d %d\n", out_in_map[40], out_in_map[13]);
    } 

    shm_map[tid] = -1;
    if (tid == 0) {
        cnt = 0;
    }
    __syncthreads();
    
    int row = bid * 64 / n_points;
    int col = bid * 64 % n_points + tid;
    int idx = -1;
    int loc = 0;
    if (row < n_points) {
        loc = out_in_map[row * n_points + col];
        if (loc != -1) {
            if (bid == 0) {
                printf("%d: %d %d %d %d\n", tid, row, col, n_points, loc);
            }
            idx = atomicAdd(&cnt, 1);
        }
    }
    __syncthreads();

    if (idx != - 1) {
        shm_map[idx] = tid;
    } 
    __syncthreads();

    loc_map[row * n_points + col] = shm_map[tid];
    
    if (tid == 0) {
        reduced_map[row * n_points / 64 + col / 64] = cdiv(cnt, 16);
    }
}

std::vector<at::Tensor> reorder_map_cuda(torch::Tensor _out_in_map) {
    /*
        assume out_in_map is [kernel_volume, n_points]
    */
    int* out_in_map = _out_in_map.data_ptr<int>();
    int n_points = _out_in_map.size(1);
    int kernel_volume = _out_in_map.size(0);

    auto options = torch::TensorOptions().dtype(_out_in_map.dtype()).device(_out_in_map.device());

    at::Tensor _loc_map = torch::zeros({kernel_volume, n_points}, options);
    int* loc_map = _loc_map.data_ptr<int>();

    at::Tensor _reduced_map = torch::zeros({kernel_volume, n_points / 64}, options);
    int* reduced_map = _reduced_map.data_ptr<int>();

    reorder_map_kernel<<<cdiv(n_points * kernel_volume, 64), 64>>>(out_in_map, loc_map, reduced_map, n_points);
    return {_loc_map, _reduced_map};
}