#include "hip/hip_runtime.h"
#include "flash_conv_cuda.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include "ptx.h"

#define cdiv(x, y) (((x) + (y) - 1) / (y))

__device__ void load_shm_A(half* shm_A, half* inputs, int* out_in_map, int kernel_size, int c_in, int ko) {
    // global layout: [128, 32]
    // shared layout: [64, 64]
    int tid = threadIdx.z * 64 + threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int row = i * 32 + tid / 4;
        int col = tid % 4 * 8;
        int row_A = out_in_map[(blockIdx.x * 128 + row) * kernel_size + (ko * 32 + col) / c_in];
        int col_A = (ko * 32 + col) % c_in;
        int shm_row = row / 2;
        int shm_col = col + (row & 1) * 32;
        shm_col = shm_col ^ ((shm_row & 3) << 3);
        if (row_A == -1) {
            *(int4*)&shm_A[shm_row * 64 + shm_col] = make_int4(0, 0, 0, 0);
        } 
        else {
            *(float4*)&shm_A[shm_row * 64 + shm_col] = *(float4*)&inputs[row_A * c_in + col_A];
        }
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [32, 72]
    int tid = threadIdx.z * 64 + threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        *(float4*)&shm_B[row * 72 + col] = *(float4*)&B[(ko * 32 + row) * N + blockIdx.y * 64 + col];
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int ki) {
    for (int m = 0; m < 4; m++) {
        int lane_id = threadIdx.x;
        int row = threadIdx.z * 64 + m * 16 + lane_id % 16;
        int col = ki * 16 + lane_id / 16 * 8;
        int shm_row = row / 2;
        int shm_col = col + (row & 1) * 32;
        shm_col = shm_col ^ ((shm_row & 3) << 3);
        uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + shm_row * 64 + shm_col);
        LDMATRIX_X4(reg_A[m * 4], reg_A[m * 4 + 1], reg_A[m * 4 + 2], reg_A[m * 4 + 3], shm_A_lane_addr);
    }
    __syncthreads();
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    for (int ni = 0; ni < 2; ni++) {
        int row = ki * 16 + lane_id % 16;
        int col = threadIdx.y * 32 + ni * 16 + lane_id / 16 * 8;
        uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 72 + col);
        LDMATRIX_X4_T(reg_B[ni * 4], reg_B[ni * 4 + 1], reg_B[ni * 4 + 2], reg_B[ni * 4 + 3], shm_B_lane_addr);
    }
}

__device__ void store_C(uint32_t* reg_C, half* C, int M, int N) {
    int lane_id = threadIdx.x;
    for (int m = 0; m < 4; m++) {
        for (int n = 0; n < 4; n++) {
            int shm_row = threadIdx.z * 64 + m * 16 + lane_id / 4;
            int shm_col = threadIdx.y * 32 + n * 8 + (lane_id % 4) * 2;
            int row = shm_row + blockIdx.x * 128;
            int col = shm_col + blockIdx.y * 64;
            C[row * N + col] = __float2half(*(float*)&reg_C[m * 16 + n * 4]);
            C[row * N + col + 1] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 1]);
            C[(row + 8) * N + col] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 2]);
            C[(row + 8) * N + col + 1] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 3]);
        }
    }
}

__global__ void flash_conv_kernel(half* inputs, half* weights, int* out_in_map, half* outputs, 
                                  int n_points, int c_in, int c_out, int kernel_size) {
    int M = n_points;
    int N = c_out;
    int K = kernel_size * c_in;
    __shared__ half shm_A[64 * 64];
    __shared__ half shm_B[32 * 72];

    uint32_t reg_A[4 * 4];
    uint32_t reg_B[4 * 2];
    uint32_t reg_C[4 * 4 * 4] = {0};

    for (int ko = 0; ko < K / 32; ko++) {
        load_shm_A(shm_A, inputs, out_in_map, kernel_size, c_in, ko);
        load_shm_B(shm_B, weights, K, N, ko);

        for (int ki = 0; ki < 2; ki++) {
            load_reg_A(reg_A, shm_A, ki);
            load_reg_B(reg_B, shm_B, ki);

            for (int m = 0; m < 4; m++) {
                for (int n = 0; n < 4; n++) {
                    int idx = m * 4 + n;
                    HMMA16816(reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3],
                            reg_A[m * 4], reg_A[m * 4 + 1], reg_A[m * 4 + 2], reg_A[m * 4 + 3],
                            reg_B[n * 2], reg_B[n * 2 + 1],
                            reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3]);
                }
            }
        }
    }
    store_C(reg_C, outputs, M, N);
}

torch::Tensor flash_conv_cuda(torch::Tensor inputs, torch::Tensor weights, torch::Tensor out_in_map) {
    int c_in = weights.size(1);
    int c_out = weights.size(2);
    int n_points = out_in_map.size(0);
    int kernel_size = out_in_map.size(1);

    auto options = torch::TensorOptions().dtype(inputs.dtype()).device(inputs.device());
    at::Tensor outputs = torch::empty({n_points, c_out}, options);

    int* out_in_map_ptr = out_in_map.data_ptr<int>();
    half* inputs_ptr = reinterpret_cast<half*>(inputs.data_ptr<at::Half>());
    half* weights_ptr = reinterpret_cast<half*>(weights.data_ptr<at::Half>());
    half* outputs_ptr = reinterpret_cast<half*>(outputs.data_ptr<at::Half>());

    dim3 num_blocks(cdiv(n_points, 128), cdiv(c_out, 64));
    dim3 num_threads(32, 2, 2);
    flash_conv_kernel<<<num_blocks, num_threads>>>(inputs_ptr, weights_ptr, out_in_map_ptr, outputs_ptr, 
                                                   n_points, c_in, c_out, kernel_size);
    return outputs;
}