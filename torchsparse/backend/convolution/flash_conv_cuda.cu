#include "hip/hip_runtime.h"
#include "flash_conv_cuda.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include "ptx.h"
#include <cuda_pipeline.h>

#define cdiv(x, y) (((x) + (y) - 1) / (y))

__device__ void load_shm_A(half* shm_A, half* inputs, int* out_in_map, int n_points, int c_in, int ko) {
    // global layout: [128, 32]
    // shared layout: [64, 64]
    int tid = threadIdx.z * 64 + threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int row = i * 32 + tid / 4;
        int col = tid % 4 * 8;
        int row_A = out_in_map[(ko * 32 + col) / c_in * n_points + blockIdx.x * 128 + row];
        int col_A = (ko * 32 + col) % c_in;
        int shm_row = row / 2;
        int shm_col = col + (row & 1) * 32;
        shm_col = shm_col ^ ((shm_row & 3) << 3);
        if (row_A == -1) {
            *(int4*)&shm_A[shm_row * 64 + shm_col] = make_int4(0, 0, 0, 0);
        } 
        else {
            __pipeline_memcpy_async(
                &shm_A[shm_row * 64 + shm_col],
                &inputs[row_A * c_in + col_A],
                16
            );
        }
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [32, 72]
    int tid = threadIdx.z * 64 + threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        __pipeline_memcpy_async(
            &shm_B[row * 72 + col],
            &B[(ko * 32 + row) * N + blockIdx.y * 64 + col],
            16
        );
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int ki) {
    for (int m = 0; m < 4; m++) {
        int lane_id = threadIdx.x;
        int row = threadIdx.z * 64 + m * 16 + lane_id % 16;
        int col = ki * 16 + lane_id / 16 * 8;
        int shm_row = row / 2;
        int shm_col = col + (row & 1) * 32;
        shm_col = shm_col ^ ((shm_row & 3) << 3);
        uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + shm_row * 64 + shm_col);
        LDMATRIX_X4(reg_A[m * 4], reg_A[m * 4 + 1], reg_A[m * 4 + 2], reg_A[m * 4 + 3], shm_A_lane_addr);
    }
    __syncthreads();
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    for (int ni = 0; ni < 2; ni++) {
        int row = ki * 16 + lane_id % 16;
        int col = threadIdx.y * 32 + ni * 16 + lane_id / 16 * 8;
        uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 72 + col);
        LDMATRIX_X4_T(reg_B[ni * 4], reg_B[ni * 4 + 1], reg_B[ni * 4 + 2], reg_B[ni * 4 + 3], shm_B_lane_addr);
    }
}

__device__ void store_C(uint32_t* reg_C, half* C, int M, int N) {
    int lane_id = threadIdx.x;
    for (int m = 0; m < 4; m++) {
        for (int n = 0; n < 4; n++) {
            int shm_row = threadIdx.z * 64 + m * 16 + lane_id / 4;
            int shm_col = threadIdx.y * 32 + n * 8 + (lane_id % 4) * 2;
            int row = shm_row + blockIdx.x * 128;
            int col = shm_col + blockIdx.y * 64;
            C[row * N + col] = __float2half(*(float*)&reg_C[m * 16 + n * 4]);
            C[row * N + col + 1] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 1]);
            C[(row + 8) * N + col] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 2]);
            C[(row + 8) * N + col + 1] = __float2half(*(float*)&reg_C[m * 16 + n * 4 + 3]);
        }
    }
}

__device__ void pipe_load(half* shm_A, half* shm_B, half* inputs, half* weights, int* out_in_map, 
                          int c_in, int c_out, int kernel_size, int n_points, int ko) {
    shm_A += (ko & 1) * 64 * 64;
    shm_B += (ko & 1) * 32 * 72;
    load_shm_A(shm_A, inputs, out_in_map, n_points, c_in, ko);
    load_shm_B(shm_B, weights, c_in * kernel_size, c_out, ko);
}

__device__ void pipe_calc(half* shm_A, half* shm_B, uint32_t* reg_A, uint32_t* reg_B, uint32_t* reg_C, int ko) {
    shm_A += (ko & 1) * 64 * 64;
    shm_B += (ko & 1) * 32 * 72;
    
    for (int ki = 0; ki < 2; ki++) {
        load_reg_A(reg_A, shm_A, ki);
        load_reg_B(reg_B, shm_B, ki);

        for (int m = 0; m < 4; m++) {
            for (int n = 0; n < 4; n++) {
                int idx = m * 4 + n;
                HMMA16816(reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3],
                        reg_A[m * 4], reg_A[m * 4 + 1], reg_A[m * 4 + 2], reg_A[m * 4 + 3],
                        reg_B[n * 2], reg_B[n * 2 + 1],
                        reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3]);
            }
        }
    }
}

__global__ void flash_conv_kernel(half* inputs, half* weights, int* out_in_map, half* outputs, 
                                  int n_points, int c_in, int c_out, int kernel_size) {
    int M = n_points;
    int N = c_out;
    int K = kernel_size * c_in;
    __shared__ half shm_A[2 * 64 * 64];
    __shared__ half shm_B[2 * 32 * 72];

    uint32_t reg_A[4 * 4];
    uint32_t reg_B[4 * 2];
    uint32_t reg_C[4 * 4 * 4] = {0};

    pipe_load(shm_A, shm_B, inputs, weights, out_in_map, c_in, c_out, kernel_size, n_points, 0);
    __pipeline_commit();

    for (int ko = 1; ko < K / 32; ko++) {
        pipe_load(shm_A, shm_B, inputs, weights, out_in_map, c_in, c_out, kernel_size, n_points, ko);
        __pipeline_commit();
        __pipeline_wait_prior(1);
        pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, ko - 1);
    }
    __pipeline_wait_prior(0);
    pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, K / 32 - 1);
    store_C(reg_C, outputs, M, N);
}

torch::Tensor flash_conv_cuda(torch::Tensor inputs, torch::Tensor weights, torch::Tensor out_in_map) {
    int c_in = weights.size(1);
    int c_out = weights.size(2);
    int n_points = out_in_map.size(1);
    int kernel_size = out_in_map.size(0);

    auto options = torch::TensorOptions().dtype(inputs.dtype()).device(inputs.device());
    at::Tensor outputs = torch::empty({n_points, c_out}, options);

    int* out_in_map_ptr = out_in_map.data_ptr<int>();
    half* inputs_ptr = reinterpret_cast<half*>(inputs.data_ptr<at::Half>());
    half* weights_ptr = reinterpret_cast<half*>(weights.data_ptr<at::Half>());
    half* outputs_ptr = reinterpret_cast<half*>(outputs.data_ptr<at::Half>());

    dim3 num_blocks(cdiv(n_points, 128), cdiv(c_out, 64));
    dim3 num_threads(32, 2, 2);
    flash_conv_kernel<<<num_blocks, num_threads>>>(inputs_ptr, weights_ptr, out_in_map_ptr, outputs_ptr, 
                                                   n_points, c_in, c_out, kernel_size);
    return outputs;
}