#include "hip/hip_runtime.h"
#include "flash_conv_sort_cuda.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include "ptx.h"
#include <cuda_pipeline.h>

namespace sparse_conv
{

#define cdiv(x, y) (((x) + (y) - 1) / (y))

namespace m64k32n64
{
__device__ void load_shm_A(half* shm_A, half* inputs, int* reorder_map, int kernel_size, int c_in, int ko) {
    // layout: [64, 32]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 32 + tid / 4;
        int col = tid % 4 * 8;
        int row_A = reorder_map[(blockIdx.x * 64 + row) * kernel_size + (ko * 32) / c_in];
        int col_A = (ko * 32 + col) % c_in;
        col = col ^ ((row & 6) << 2);
        if (row_A == -1) {
            *(int4*)&shm_A[row * 32 + col] = make_int4(0, 0, 0, 0);
        } 
        else {
            __pipeline_memcpy_async(
                &shm_A[row * 32 + col],
                &inputs[row_A * c_in + col_A],
                16
            );
        }
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [32, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        int shm_col = col ^ ((row & 7) << 3);
        __pipeline_memcpy_async(
            &shm_B[row * 64 + shm_col],
            &B[(ko * 32 + row) * N + blockIdx.y * 64 + col],
            16
        );
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int ki, int m) {
    int lane_id = threadIdx.x;
    int row = m * 16 + lane_id % 16;
    int col = ki * 16 + lane_id / 16 * 8;
    col = col ^ ((row & 6) << 2);
    uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + row * 32 + col);
    LDMATRIX_X4(reg_A[ki * 16 + m * 4], reg_A[ki * 16 + m * 4 + 1], reg_A[ki * 16 + m * 4 + 2], reg_A[ki * 16 + m * 4 + 3], shm_A_lane_addr);
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    int row = ki * 16 + lane_id % 16;
    int col = threadIdx.y * 16  + lane_id / 16 * 8;
    col = col ^ ((row & 7) << 3);
    uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 64 + col);
    LDMATRIX_X4_T(reg_B[ki * 4], reg_B[ki * 4 + 1], reg_B[ki * 4 + 2], reg_B[ki * 4 + 3], shm_B_lane_addr);
}

__device__ void store_C(uint32_t* reg_C, half* C, int* reorder_loc, int M, int N) {
    int lane_id = threadIdx.x;
    for (int m = 0; m < 4; m++) {
        for (int n = 0; n < 2; n++) {
            int shm_row = m * 16 + lane_id / 4;
            int shm_col = threadIdx.y * 16 + n * 8 + (lane_id % 4) * 2;
            int row = shm_row + blockIdx.x * 64;
            int col = shm_col + blockIdx.y * 64;
            int row_8 = reorder_loc[row + 8];
            row = reorder_loc[row];
            if (row < M) {
                C[row * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4]);
                C[row * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 1]);
            }
            if (row_8 < M) {
                C[row_8 * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 2]);
                C[row_8 * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 3]);
            }
        }
    }
}

__device__ void pipe_load(half* shm_A, half* shm_B, half* inputs, half* weights, int* reorder_map, 
                          int kernel_size, int c_in, int N, int ko, int loc) {
    shm_A += loc * 64 * 32;
    shm_B += loc * 32 * 64;
    load_shm_A(shm_A, inputs, reorder_map, kernel_size, c_in, ko);
    load_shm_B(shm_B, weights, kernel_size * c_in, N, ko);
}

__device__ void pipe_calc(half* shm_A, half* shm_B, uint32_t* reg_A, uint32_t* reg_B, uint32_t* reg_C, int mma_flag, int ko, int loc) {
    shm_A += loc * 64 * 32;
    shm_B += loc * 32 * 64;
    
    for (int ki = 0; ki < 2; ki++) {
        load_reg_B(reg_B, shm_B, ki);
    }

    for (int m = 0; m < 4; m++) {
        if (mma_flag & (1 << m)) {
            for (int ki = 0; ki < 2; ki++) {
                load_reg_A(reg_A, shm_A, ki, m);
            }
            for (int ki = 0; ki < 2; ki++) {
                for (int n = 0; n < 2; n++) {
                    int idx = m * 2 + n;
                    HMMA16816(reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3],
                        reg_A[ki * 16 + m * 4], reg_A[ki * 16 + m * 4 + 1], reg_A[ki * 16 + m * 4 + 2], reg_A[ki * 16 + m * 4 + 3],
                        reg_B[ki * 4 + n * 2], reg_B[ki * 4 + n * 2 + 1],
                        reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3]);
                }
            }
        }
    }
}

__global__ void sparse_conv_kernel(half* inputs, half* weights, int* reorder_map, int* reduced_mask, int* mma_mask,
                                       int* reorder_loc, half* outputs, 
                                       int n_points, int c_in, int c_out, int kernel_size) {
    int M = n_points;
    int N = c_out;
    int K = kernel_size * c_in;
    __shared__ half shm_A[2 * 64 * 32];
    __shared__ half shm_B[2 * 32 * 64];

    uint32_t reg_A[2 * 4 * 4];
    uint32_t reg_B[2 * 2 * 2];
    uint32_t reg_C[8 * 2 * 4] = {0};

    int ko;
    int idx0, idx1;
    int loc0, loc1;

    for (ko = 0; ko < K / 32; ko++) {
        bool flag = reduced_mask[blockIdx.x] & (1 << (ko * 32 / c_in));
        if (flag) {
            idx1 = ko;
            loc1 = 0;
            pipe_load(shm_A, shm_B, inputs, weights, reorder_map, kernel_size, c_in, N, idx1, loc1);
            __pipeline_commit();
            break;
        }
    }

    if (ko < K / 32) {
        for (ko += 1; ko < K / 32; ko++) {
            bool flag = reduced_mask[blockIdx.x] & (1 << (ko * 32 / c_in));
            if (flag) {
                idx0 = idx1;
                loc0 = loc1;
                idx1 = ko;
                loc1 = loc0 ^ 1;
                int mma_flag = 0;
                for (int i = 0; i < 4; i++) {
                    if (mma_mask[blockIdx.x * 4 + i] & (1 << (idx0 * 32 / c_in))) {
                        mma_flag = mma_flag + (1 << i); 
                    }
                }
                pipe_load(shm_A, shm_B, inputs, weights, reorder_map, kernel_size, c_in, N, idx1, loc1);
                __pipeline_commit();
                __pipeline_wait_prior(1);
                pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, mma_flag, idx0, loc0);
                __syncthreads();
            }
        }

        __pipeline_wait_prior(0);
        int mma_flag = 0;
        for (int i = 0; i < 4; i++) {
            if (mma_mask[blockIdx.x * 4 + i] & (1 << (idx1 * 32 / c_in))) {
                mma_flag = mma_flag + (1 << i); 
            }
        }
        pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, mma_flag, idx1, loc1);
        __syncthreads();
    }

    store_C(reg_C, outputs, reorder_loc, M, N);
}
}


namespace m128k32n64
{
__device__ void load_shm_A(half* shm_A, half* inputs, int* reorder_map, int kernel_size, int c_in, int ko) {
    // global layout: [128, 32]
    // shared layout: [64, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int row = i * 32 + tid / 4;
        int col = tid % 4 * 8;
        int row_A = reorder_map[(blockIdx.x * 128 + row) * kernel_size + (ko * 32) / c_in];
        int col_A = (ko * 32 + col) % c_in;
        // int shm_row = row / 2;
        // int shm_col = col + (row & 1) * 32;
        // shm_col = shm_col ^ ((shm_row & 7) << 3);
        col = col ^ ((row & 6) << 2);
        if (row_A == -1) {
            // *(int4*)&shm_A[shm_row * 64 + shm_col] = make_int4(0, 0, 0, 0);
            *(int4*)&shm_A[row * 32 + col] = make_int4(0, 0, 0, 0);
        } 
        else {
            __pipeline_memcpy_async(
                // &shm_A[shm_row * 64 + shm_col],
                &shm_A[row * 32 + col],
                &inputs[row_A * c_in + col_A],
                16
            );
        }
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [32, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 2; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        int shm_col = col ^ ((row & 7) << 3);
        __pipeline_memcpy_async(
            &shm_B[row * 64 + shm_col],
            &B[(ko * 32 + row) * N + blockIdx.y * 64 + col],
            16
        );
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int ki, int m) {
    // for (int m = 0; m < 8; m++) {
        int lane_id = threadIdx.x;
        int row = m * 16 + lane_id % 16;
        int col = ki * 16 + lane_id / 16 * 8;
        // int shm_row = row / 2;
        // int shm_col = col + (row & 1) * 32;
        // shm_col = shm_col ^ ((shm_row & 7) << 3);
        col = col ^ ((row & 6) << 2);
        // uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + shm_row * 64 + shm_col);
        uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + row * 32 + col);
        LDMATRIX_X4(reg_A[ki * 32 + m * 4], reg_A[ki * 32 + m * 4 + 1], reg_A[ki * 32 + m * 4 + 2], reg_A[ki * 32 + m * 4 + 3], shm_A_lane_addr);
    // }
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    int row = ki * 16 + lane_id % 16;
    int col = threadIdx.y * 16  + lane_id / 16 * 8;
    col = col ^ ((row & 7) << 3);
    uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 64 + col);
    LDMATRIX_X4_T(reg_B[ki * 4], reg_B[ki * 4 + 1], reg_B[ki * 4 + 2], reg_B[ki * 4 + 3], shm_B_lane_addr);
}

__device__ void store_C(uint32_t* reg_C, half* C, int* reorder_loc, int M, int N) {
    int lane_id = threadIdx.x;
    for (int m = 0; m < 8; m++) {
        for (int n = 0; n < 2; n++) {
            int shm_row = m * 16 + lane_id / 4;
            int shm_col = threadIdx.y * 16 + n * 8 + (lane_id % 4) * 2;
            int row = shm_row + blockIdx.x * 128;
            int col = shm_col + blockIdx.y * 64;
            int row_8 = reorder_loc[row + 8];
            row = reorder_loc[row];
            if (row < M) {
                C[row * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4]);
                C[row * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 1]);
            }
            if (row_8 < M) {
                C[row_8 * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 2]);
                C[row_8 * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 3]);
            }
        }
    }
}

__device__ void pipe_load(half* shm_A, half* shm_B, half* inputs, half* weights, int* reorder_map, 
                          int kernel_size, int c_in, int N, int ko, int loc) {
    shm_A += loc * 128 * 32;
    shm_B += loc * 32 * 64;
    load_shm_A(shm_A, inputs, reorder_map, kernel_size, c_in, ko);
    load_shm_B(shm_B, weights, kernel_size * c_in, N, ko);
}

__device__ void pipe_calc(half* shm_A, half* shm_B, uint32_t* reg_A, uint32_t* reg_B, uint32_t* reg_C, int mma_flag, int ko, int loc) {
    shm_A += loc * 128 * 32;
    shm_B += loc * 32 * 64;
    
    for (int ki = 0; ki < 2; ki++) {
        load_reg_B(reg_B, shm_B, ki);
    }

    for (int m = 0; m < 8; m++) {
        if (mma_flag & (1 << m)) {
            for (int ki = 0; ki < 2; ki++) {
                load_reg_A(reg_A, shm_A, ki, m);
            }
            for (int ki = 0; ki < 2; ki++) {
                for (int n = 0; n < 2; n++) {
                    int idx = m * 2 + n;
                    HMMA16816(reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3],
                        reg_A[ki * 32 + m * 4], reg_A[ki * 32 + m * 4 + 1], reg_A[ki * 32 + m * 4 + 2], reg_A[ki * 32 + m * 4 + 3],
                        reg_B[ki * 4 + n * 2], reg_B[ki * 4 + n * 2 + 1],
                        reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3]);
                }
            }
        }
    }
}

__global__ void sparse_conv_kernel(half* inputs, half* weights, int* reorder_map, int* reduced_mask, int* mma_mask,
                                       int* reorder_loc, half* outputs, 
                                       int n_points, int c_in, int c_out, int kernel_size) {
    int M = n_points;
    int N = c_out;
    int K = kernel_size * c_in;
    __shared__ half shm_A[2 * 128 * 32];
    __shared__ half shm_B[2 * 32 * 64];

    uint32_t reg_A[2 * 8 * 4];
    uint32_t reg_B[2 * 2 * 2];
    uint32_t reg_C[8 * 2 * 4] = {0};

    int ko;
    int idx0, idx1;
    int loc0, loc1;

    for (ko = 0; ko < K / 32; ko++) {
        bool flag = reduced_mask[blockIdx.x] & (1 << (ko * 32 / c_in));
        if (flag) {
            idx1 = ko;
            loc1 = 0;
            pipe_load(shm_A, shm_B, inputs, weights, reorder_map, kernel_size, c_in, N, idx1, loc1);
            __pipeline_commit();
            break;
        }
    }

    if (ko < K / 32) {
        for (ko += 1; ko < K / 32; ko++) {
            bool flag = reduced_mask[blockIdx.x] & (1 << (ko * 32 / c_in));
            if (flag) {
                idx0 = idx1;
                loc0 = loc1;
                idx1 = ko;
                loc1 = loc0 ^ 1;
                int mma_flag = 0;
                for (int i = 0; i < 8; i++) {
                    if (mma_mask[blockIdx.x * 8 + i] & (1 << (idx0 * 32 / c_in))) {
                        mma_flag = mma_flag + (1 << i); 
                    }
                }
                pipe_load(shm_A, shm_B, inputs, weights, reorder_map, kernel_size, c_in, N, idx1, loc1);
                __pipeline_commit();
                __pipeline_wait_prior(1);
                pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, mma_flag, idx0, loc0);
                __syncthreads();
            }
        }

        __pipeline_wait_prior(0);
        int mma_flag = 0;
        for (int i = 0; i < 8; i++) {
            if (mma_mask[blockIdx.x * 8 + i] & (1 << (idx1 * 32 / c_in))) {
                mma_flag = mma_flag + (1 << i); 
            }
        }
        pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, mma_flag, idx1, loc1);
        __syncthreads();
    }

    store_C(reg_C, outputs, reorder_loc, M, N);
}
}

namespace m128k64n64
{
__device__ void load_shm_A(half* shm_A, half* inputs, int* reorder_map, int kernel_size, int c_in, int ko) {
    // layout: [128, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 8; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        int row_A = reorder_map[(blockIdx.x * 128 + row) * kernel_size + (ko * 64) / c_in];
        int col_A = (ko * 64 + col) % c_in;
        int shm_row = row;
        int shm_col = col ^ ((shm_row & 7) << 3);
        if (row_A == -1) {
            *(int4*)&shm_A[shm_row * 64 + shm_col] = make_int4(0, 0, 0, 0);
        } 
        else {
            __pipeline_memcpy_async(
                &shm_A[shm_row * 64 + shm_col],
                &inputs[row_A * c_in + col_A],
                16
            );
        }
    }
    __syncthreads();
}

__device__ void load_shm_B(half* shm_B, half* B, int K, int N, int ko) {
    // layout: [64, 64]
    int tid = threadIdx.y * 32 + threadIdx.x;
    for (int i = 0; i < 4; i++) {
        int row = i * 16 + tid / 8;
        int col = tid % 8 * 8;
        int shm_col = col ^ ((row & 7) << 3);
        __pipeline_memcpy_async(
            &shm_B[row * 64 + shm_col],
            &B[(ko * 64 + row) * N + blockIdx.y * 64 + col],
            16
        );
    }
    __syncthreads();
}

__device__ void load_reg_A(uint32_t* reg_A, half* shm_A, int ki, int m) {
    // for (int m = 0; m < 8; m++) {
        int lane_id = threadIdx.x;
        int row = m * 16 + lane_id % 16;
        int col = ki * 16 + lane_id / 16 * 8;
        int shm_row = row;
        int shm_col = col ^ ((shm_row & 7) << 3);
        uint32_t shm_A_lane_addr = __cvta_generic_to_shared(shm_A + shm_row * 64 + shm_col);
        LDMATRIX_X4(reg_A[ki * 32 + m * 4], reg_A[ki * 32 + m * 4 + 1], reg_A[ki * 32 + m * 4 + 2], reg_A[ki * 32 + m * 4 + 3], shm_A_lane_addr);
    // }
}

__device__ void load_reg_B(uint32_t* reg_B, half* shm_B, int ki) {
    int lane_id = threadIdx.x;
    int row = ki * 16 + lane_id % 16;
    int col = threadIdx.y * 16  + lane_id / 16 * 8;
    col = col ^ ((row & 7) << 3);
    uint32_t shm_B_lane_addr = __cvta_generic_to_shared(shm_B + row * 64 + col);
    LDMATRIX_X4_T(reg_B[ki * 4], reg_B[ki * 4 + 1], reg_B[ki * 4 + 2], reg_B[ki * 4 + 3], shm_B_lane_addr);
}

__device__ void store_C(uint32_t* reg_C, half* C, int* reorder_loc, int M, int N) {
    int lane_id = threadIdx.x;
    for (int m = 0; m < 8; m++) {
        for (int n = 0; n < 2; n++) {
            int shm_row = m * 16 + lane_id / 4;
            int shm_col = threadIdx.y * 16 + n * 8 + (lane_id % 4) * 2;
            int row = shm_row + blockIdx.x * 128;
            int col = shm_col + blockIdx.y * 64;
            int row_8 = reorder_loc[row + 8];
            row = reorder_loc[row];
            if (row < M) {
                C[row * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4]);
                C[row * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 1]);
            }
            if (row_8 < M) {
                C[row_8 * N + col] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 2]);
                C[row_8 * N + col + 1] = __float2half(*(float*)&reg_C[m * 8 + n * 4 + 3]);
            }
        }
    }
}

__device__ void pipe_load(half* shm_A, half* shm_B, half* inputs, half* weights, int* reorder_map, 
                          int kernel_size, int c_in, int N, int ko) {
    load_shm_A(shm_A, inputs, reorder_map, kernel_size, c_in, ko);
    load_shm_B(shm_B, weights, kernel_size * c_in, N, ko);
}

__device__ void pipe_calc(half* shm_A, half* shm_B, uint32_t* reg_A, uint32_t* reg_B, uint32_t* reg_C, int mma_flag) {
    for (int ki = 0; ki < 4; ki++) {
        load_reg_B(reg_B, shm_B, ki);
    }

    for (int m = 0; m < 8; m++) {
        if (mma_flag & (1 << m)) {
            for (int ki = 0; ki < 4l; ki++) {
                load_reg_A(reg_A, shm_A, ki, m);
            }
            for (int ki = 0; ki < 4; ki++) {
                for (int n = 0; n < 2; n++) {
                    int idx = m * 2 + n;
                    HMMA16816(reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3],
                        reg_A[ki * 32 + m * 4], reg_A[ki * 32 + m * 4 + 1], reg_A[ki * 32 + m * 4 + 2], reg_A[ki * 32 + m * 4 + 3],
                        reg_B[ki * 4 + n * 2], reg_B[ki * 4 + n * 2 + 1],
                        reg_C[idx * 4], reg_C[idx * 4 + 1], reg_C[idx * 4 + 2], reg_C[idx * 4 + 3]);
                }
            }
        }
    }
}

__global__ void sparse_conv_kernel(half* inputs, half* weights, int* reorder_map, int* reduced_mask, int* mma_mask,
                                       int* reorder_loc, half* outputs, 
                                       int n_points, int c_in, int c_out, int kernel_size) {
    int M = n_points;
    int N = c_out;
    int K = kernel_size * c_in;
    __shared__ half shm_A[128 * 64];
    __shared__ half shm_B[64 * 64];

    uint32_t reg_A[4 * 8 * 4];
    uint32_t reg_B[4 * 2 * 2];
    uint32_t reg_C[8 * 2 * 4] = {0};

    for (int ko = 0; ko < K / 64; ko++) {
        bool flag = reduced_mask[blockIdx.x] & (1 << (ko * 64 / c_in));
        if (flag) {
            int mma_flag = 0;
            for (int i = 0; i < 8; i++) {
                if (mma_mask[blockIdx.x * 8 + i] & (1 << (ko * 64 / c_in))) {
                    mma_flag = mma_flag + (1 << i); 
                }
            }
            pipe_load(shm_A, shm_B, inputs, weights, reorder_map, kernel_size, c_in, N, ko);
            __pipeline_commit();
            __pipeline_wait_prior(0);
            pipe_calc(shm_A, shm_B, reg_A, reg_B, reg_C, mma_flag);
            __syncthreads();
        }
    }
    store_C(reg_C, outputs, reorder_loc, M, N);
}
}

}

torch::Tensor sparse_conv_cuda(torch::Tensor inputs, torch::Tensor weights, torch::Tensor reorder_map, 
                               torch::Tensor reduced_mask, torch::Tensor mma_mask, torch::Tensor reorder_loc, 
                               int num_out_feats) {
    int c_in = weights.size(1);
    int c_out = weights.size(2);
    int n_points = num_out_feats;
    int kernel_size = reorder_map.size(1);

    auto options = torch::TensorOptions().dtype(inputs.dtype()).device(inputs.device());
    at::Tensor outputs = torch::empty({n_points, c_out}, options);

    int* reorder_map_ptr = reorder_map.data_ptr<int>();
    int* reduced_mask_ptr = reduced_mask.data_ptr<int>();
    int* mma_mask_ptr = mma_mask.data_ptr<int>();
    int* reorder_loc_ptr = reorder_loc.data_ptr<int>();
    half* inputs_ptr = reinterpret_cast<half*>(inputs.data_ptr<at::Half>());
    half* weights_ptr = reinterpret_cast<half*>(weights.data_ptr<at::Half>());
    half* outputs_ptr = reinterpret_cast<half*>(outputs.data_ptr<at::Half>());

    dim3 num_blocks(cdiv(n_points, 128), cdiv(c_out, 64));
    dim3 num_threads(32, 4);
    sparse_conv::m128k64n64::sparse_conv_kernel<<<num_blocks, num_threads>>>
                (inputs_ptr, weights_ptr, reorder_map_ptr, reduced_mask_ptr, mma_mask_ptr, reorder_loc_ptr,
                outputs_ptr, n_points, c_in, c_out, kernel_size);

    // dim3 num_blocks(cdiv(n_points, 64), cdiv(c_out, 64));
    // dim3 num_threads(32, 4);
    // sparse_conv::m64k32n64::sparse_conv_kernel<<<num_blocks, num_threads>>>
    //             (inputs_ptr, weights_ptr, reorder_map_ptr, reduced_mask_ptr, mma_mask_ptr, reorder_loc_ptr,
    //             outputs_ptr, n_points, c_in, c_out, kernel_size);
    return outputs;
}