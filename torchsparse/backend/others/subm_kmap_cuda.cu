#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "subm_kmap_cuda.h"
#include <hip/hip_fp16.h>

#include <cuda/std/tuple>
#include <cub/device/device_radix_sort.cuh>

#include <bitset>
#include <cstdint>
#include <functional>
#include <limits>
#include <type_traits>
#include "cub/block/radix_rank_sort_operations.cuh"

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <ctime>
#include <sys/time.h>

struct custom_t
{
  int loc, x, y, z, w;
};

struct decomposer_t
{
  __host__ __device__ //
  ::cuda::std::tuple<int&, int&, int&, int&> operator()(custom_t &key) const
  {
    return {key.x, key.y, key.z, key.w};
  }
};

#define CDIV(X, Y) (((X) + (Y) - 1) / (Y))  
#define INT4(X, Y, Z) make_int4(0, X, Y, Z)
#define COORD(tid) (make_int4(coords[tid].x, coords[tid].y, coords[tid].z, coords[tid].w))

__device__ int4 add(int4 a, int4 b) {
    return make_int4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ bool great_equal(int4 a, int4 b) {
    if (a.x > b.x) return true;
    else if (a.x < b.x) return false;
    if (a.y > b.y) return true;
    else if (a.y < b.y) return false;
    if (a.z > b.z) return true;
    else if (a.z < b.z) return false;
    if (a.w > b.w) return true;
    else if (a.w < b.w) return false;
    return true;
}

__device__ bool equal(int4 a, int4 b) {
    if (a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w) {
        return true;
    } else {
        return false;
    }
}

__global__ void subm_kmap_kernel(
        custom_t* coords, 
        int* out_in_map, 
        int n_points) {
    int4 offsets[14] = {
        INT4(0, 0, 0), INT4(0, 0, 1), INT4(0, 1, -1), INT4(0, 1, 0), INT4(0, 1, 1), INT4(1, -1, -1), INT4(1, -1, 0), 
        INT4(1, -1, 1), INT4(1, 0, -1), INT4(1, 0, 0), INT4(1, 0, 1), INT4(1, 1, -1), INT4(1, 1, 0), INT4(1, 1, 1)
    };

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n_points) {
        int loc = coords[tid].loc;
        out_in_map[loc * 27 + 27 / 2] = loc;
        int4 out_coord = COORD(tid);

        int l = tid, r = n_points - 1, mid;
        for (int i = 1; i <= 27 / 2; i++) {
            int4 in_coord = add(out_coord, offsets[i]); 
            l = tid;
            r = n_points - 1;
            while (l < r) {
                mid = (l + r) >> 1;
                if (great_equal(COORD(mid), in_coord)) {
                    r = mid;
                } else {
                    l = mid + 1;
                }
            }
            r = l;
            if (equal(COORD(r), in_coord)) {
                int loc_r = coords[r].loc;
                int idx = (offsets[i].w + 1) * 9 + (offsets[i].z + 1) * 3 + (offsets[i].y + 1);
                out_in_map[loc * 27 + idx] = loc_r;
                out_in_map[loc_r * 27 + 26 - idx] = loc;
            } 
        }
    }
}

__global__ void data_copy(int4* coords, custom_t* loc_coords, int n_points) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n_points) {
        int4 coord = coords[tid];
        loc_coords[tid] = {tid, coord.x, coord.y, coord.z, coord.w};
    }
}

at::Tensor subm_kmap_cuda(at::Tensor _coords, at::Tensor _kernel_sizes) {
    struct timeval tv;
    double start, end;
    int n_points = _coords.size(0);
    int4 *coords = (int4*)_coords.data_ptr<int>();
    custom_t* loc_coords;
    custom_t* sorted_coords;

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    hipMalloc(&loc_coords, n_points * sizeof(custom_t));
    hipMalloc(&sorted_coords, n_points * sizeof(custom_t));
    data_copy<<<CDIV(n_points, 256), 256>>>(coords, loc_coords, n_points);
    hipDeviceSynchronize();

    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("time1: %lf\n", end - start);
    
    std::uint8_t *d_temp_storage{};
    std::size_t temp_storage_bytes{};

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;

    hipcub::DeviceRadixSort::SortKeys(d_temp_storage,
                                 temp_storage_bytes,
                                 loc_coords,
                                 sorted_coords,
                                 n_points,
                                 decomposer_t{});
    
    thrust::device_vector<std::uint8_t> temp_storage(temp_storage_bytes);
    d_temp_storage = thrust::raw_pointer_cast(temp_storage.data());

    hipcub::DeviceRadixSort::SortKeys(d_temp_storage,
                                 temp_storage_bytes,
                                 loc_coords,
                                 sorted_coords,
                                 n_points,
                                 decomposer_t{});
    hipDeviceSynchronize();

    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("time2: %lf\n", end - start);

    auto options = torch::TensorOptions()
                    .dtype(at::ScalarType::Int)
                    .device(_coords.device());

    at::Tensor _out_in_map = torch::full({CDIV(n_points, 128) * 128, 27}, -1, options);
    int* out_in_map = _out_in_map.data_ptr<int>();

    gettimeofday(&tv, nullptr);
    start = tv.tv_sec + tv.tv_usec / 1.0e6;
    
    subm_kmap_kernel<<<CDIV(n_points, 256), 256>>>(
        sorted_coords,
        out_in_map, 
        n_points);
    hipDeviceSynchronize();

    gettimeofday(&tv, nullptr);
    end = tv.tv_sec + tv.tv_usec / 1.0e6;
    printf("time3: %lf\n", end - start);

    return _out_in_map;
}